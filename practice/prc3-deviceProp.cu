// sample code for getting devices info (using CudaGetDeviceCount, cudaGetDeviceProperties)
// https://cpp.hotexamples.com/examples/-/-/cudaGetDeviceCount/cpp-cudagetdevicecount-function-examples.html


#include <hip/hip_runtime.h>
#include <stdio.h>

// Host code
int main() {
    // get device count
    int deviceCount = 0;
    hipError_t err_id = hipGetDeviceCount(&deviceCount);
    if (err_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int) err_id, hipGetErrorString(err_id));
        return false;
    }
    printf("\ndevice count: %d\n", deviceCount);

    // get device properties
    int dev, driverVersion = 0, runtimeVersion = 0;

    for (dev = 0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

        // Console log
        hipDriverGetVersion(&driverVersion);
        hipRuntimeGetVersion(&runtimeVersion);
        printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion / 1000,
               (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
        printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);
    }
    return 0;
}

/*
   device count: 2

   Device 0: "TITAN V"
    CUDA Driver Version / Runtime Version          11.0 / 11.0
    CUDA Capability Major/Minor version number:    7.0

   Device 1: "TITAN V"
    CUDA Driver Version / Runtime Version          11.0 / 11.0
    CUDA Capability Major/Minor version number:    7.0
 */
